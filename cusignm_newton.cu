#include "hip/hip_runtime.h"
/* MIT License
 *
 * Copyright (c) 2024 Maximilian Behr
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <thrust/device_ptr.h>
#include <thrust/inner_product.h>

#include "checkcuda.h"
#include "cusignm.h"
#include "cusignm_frobenius.h"
#include "cusignm_traits.h"

const static hipsolverAlgMode_t CUSOLVER_ALG = HIPSOLVER_ALG_0;

template <typename T>
static int cusignm_NewtonBufferSize(const int n, size_t *d_bufferSize, size_t *h_bufferSize) {
    /*-----------------------------------------------------------------------------
     * initialize with zero
     *-----------------------------------------------------------------------------*/
    *d_bufferSize = 0;
    *h_bufferSize = 0;

    /*-----------------------------------------------------------------------------
     * get device and host workspace size for LU factorization
     *-----------------------------------------------------------------------------*/
    // create cusolver handle
    hipsolverHandle_t cusolverH;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    // create cusolver params
    hipsolverDnParams_t params;
    CHECK_CUSOLVER(hipsolverDnCreateParams(&params));
    CHECK_CUSOLVER(hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, CUSOLVER_ALG));

    // compute workspace size
    CHECK_CUSOLVER(hipsolverDnXgetrf_bufferSize(cusolverH, params, n, n, cusignm_traits<T>::dataType, nullptr, n, cusignm_traits<T>::computeType, d_bufferSize, h_bufferSize));

    // free workspace
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    CHECK_CUSOLVER(hipsolverDnDestroyParams(params));

    /*-----------------------------------------------------------------------------
     * compute final workspace size
     *-----------------------------------------------------------------------------*/
    *d_bufferSize += sizeof(T) * n * n * 3 + sizeof(int64_t) * n + sizeof(int);

    return 0;
}

int cusignm_sNewtonBufferSize(const int n, size_t *d_bufferSize, size_t *h_bufferSize) {
    return cusignm_NewtonBufferSize<float>(n, d_bufferSize, h_bufferSize);
}

int cusignm_dNewtonBufferSize(const int n, size_t *d_bufferSize, size_t *h_bufferSize) {
    return cusignm_NewtonBufferSize<double>(n, d_bufferSize, h_bufferSize);
}

int cusignm_cNewtonBufferSize(const int n, size_t *d_bufferSize, size_t *h_bufferSize) {
    return cusignm_NewtonBufferSize<hipComplex>(n, d_bufferSize, h_bufferSize);
}

int cusignm_zNewtonBufferSize(const int n, size_t *d_bufferSize, size_t *h_bufferSize) {
    return cusignm_NewtonBufferSize<hipDoubleComplex>(n, d_bufferSize, h_bufferSize);
}

template <typename T>
__global__ void identity(const int n, T *A, const int lda) {
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    int j0 = blockIdx.y * blockDim.y + threadIdx.y;
    for (int j = j0; j < n; j += gridDim.y * blockDim.y) {
        for (int i = i0; i < n; i += gridDim.x * blockDim.x) {
            A[i + j * lda] = (i == j) ? cusignm_traits<T>::one : cusignm_traits<T>::zero;
        }
    }
}

template <typename T>
static int cusignm_Newton(const int n, const T *A, void *d_buffer, void *h_buffer, T *S) {
    /*-----------------------------------------------------------------------------
     * derived types
     *-----------------------------------------------------------------------------*/
    using Scalar = typename cusignm_traits<T>::S;  // real type: double for hipDoubleComplex, float for hipComplex

    /*-----------------------------------------------------------------------------
     * constants and variables
     *-----------------------------------------------------------------------------*/
    int ret = 0, iter = 1;
    constexpr int maxiter = 100;
    const Scalar tol = std::sqrt(std::numeric_limits<Scalar>::epsilon());  // square root of machine epsilon - newton iteration converges quadratically
    Scalar alpha, beta, mu;

    /*-----------------------------------------------------------------------------
     * create cuBlas handle
     *-----------------------------------------------------------------------------*/
    hipblasHandle_t cublasH;
    CHECK_CUBLAS(hipblasCreate(&cublasH));

    /*-----------------------------------------------------------------------------
     * create cusolver handle and params structure
     *-----------------------------------------------------------------------------*/
    hipsolverHandle_t cusolverH;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));

    hipsolverDnParams_t params;
    CHECK_CUSOLVER(hipsolverDnCreateParams(&params));
    CHECK_CUSOLVER(hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, CUSOLVER_ALG));

    /*-----------------------------------------------------------------------------
     * split memory buffer
     * memory layout: |Sold, Stmp, SoldInv, ipiv, info, d_work|
     *-----------------------------------------------------------------------------*/
    T *Sold = reinterpret_cast<T *>(d_buffer);
    T *Stmp = reinterpret_cast<T *>(Sold + n * n);                   // put Stmp after Sold
    T *SoldInv = reinterpret_cast<T *>(Stmp + n * n);                // put SoldInv after Sold
    int64_t *d_ipiv = reinterpret_cast<int64_t *>(SoldInv + n * n);  // put d_ipiv after SoldInv
    int *d_info = reinterpret_cast<int *>(d_ipiv + n);               // put d_info after d_ipiv
    void *d_work = reinterpret_cast<int *>(d_info + 1);              // put d_work after d_info
    void *h_work = reinterpret_cast<void *>(h_buffer);
    std::swap(S, Sold);

    /*-----------------------------------------------------------------------------
     * copy A to Sold
     *-----------------------------------------------------------------------------*/
    CHECK_CUDA(hipMemcpy(Sold, A, sizeof(T) * n * n, hipMemcpyDeviceToDevice));

    /*-----------------------------------------------------------------------------
     * newton iteration
     *-----------------------------------------------------------------------------*/
    iter = 1;
    static_assert(maxiter >= 1, "maxiter >= 1");
    while (true) {
        /*-----------------------------------------------------------------------------
         * copy Sold to Stmp
         *-----------------------------------------------------------------------------*/
        CHECK_CUDA(hipMemcpy(Stmp, Sold, sizeof(T) * n * n, hipMemcpyDeviceToDevice));

        /*-----------------------------------------------------------------------------
         * compute inv(S)^H
         *-----------------------------------------------------------------------------*/
        // workspace query for LU factorization
        size_t lworkdevice = 0, lworkhost = 0;
        CHECK_CUSOLVER(hipsolverDnXgetrf_bufferSize(cusolverH, params, n, n, cusignm_traits<T>::dataType, Stmp, n, cusignm_traits<T>::computeType, &lworkdevice, &lworkhost));

        // compute LU factorization and set right side to identity on different streams
        hipStream_t streamLU, streamIdentity;
        CHECK_CUDA(hipStreamCreate(&streamLU));
        CHECK_CUDA(hipStreamCreate(&streamIdentity));
        CHECK_CUSOLVER(hipsolverSetStream(cusolverH, streamLU));
        CHECK_CUSOLVER(hipsolverDnXgetrf(cusolverH, params, n, n, cusignm_traits<T>::dataType, Stmp, n, d_ipiv, cusignm_traits<T>::computeType, d_work, lworkdevice, h_work, lworkhost, d_info));

        // set right-hand side to identity
        {
            dim3 grid((n + 15) / 16, (n + 15) / 16);
            dim3 block(16, 16);
            identity<<<grid, block, 0, streamIdentity>>>(n, SoldInv, n);
            CHECK_CUDA(hipPeekAtLastError());
        }

        // synchronize and destroy streams
        CHECK_CUDA(hipStreamSynchronize(streamLU));
        CHECK_CUDA(hipStreamSynchronize(streamIdentity));
        CHECK_CUDA(hipStreamDestroy(streamLU));
        CHECK_CUDA(hipStreamDestroy(streamIdentity));
        CHECK_CUSOLVER(hipsolverSetStream(cusolverH, 0));

        // solve the linear system to compute the hermitian/transposed inverse of Sold
        CHECK_CUSOLVER(hipsolverDnXgetrs(cusolverH, params, HIPBLAS_OP_N, n, n, cusignm_traits<T>::dataType, Stmp, n, d_ipiv, cusignm_traits<T>::computeType, SoldInv, n, d_info));

        /*-----------------------------------------------------------------------------
         * compute alpha and beta to compute mu for the first iteration
         *-----------------------------------------------------------------------------*/
        if (iter == 1) {
            CHECK_CUSIGNM(cusignm_normFro(n, n, A, &alpha));
            CHECK_CUSIGNM(cusignm_normFro(n, n, SoldInv, &beta));
            beta = Scalar{1.0} / beta;
            mu = Scalar{1.0} / std::sqrt(alpha * beta);
        }

        /*-----------------------------------------------------------------------------
         * update S as 0.5 * (mu * S + 1/mu * inv(S))
         *-----------------------------------------------------------------------------*/
        {
            T a, b;
            if constexpr (std::is_same<T, hipComplex>::value || std::is_same<T, hipDoubleComplex>::value) {
                a = T{Scalar{0.5} * mu, Scalar{0.0}}, b = T{Scalar{0.5} / mu, Scalar{0.0}};
            } else {
                a = Scalar{0.5} * mu, b = Scalar{0.5} / mu;
            }
            CHECK_CUBLAS(cusignm_traits<T>::cublasXgeam(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &a, Sold, n, &b, SoldInv, n, S, n));
        }

        /*-----------------------------------------------------------------------------
         * update mu for the next iteration
         *-----------------------------------------------------------------------------*/
        if (iter == 1) {
            mu = std::sqrt(Scalar{2.0} * std::sqrt(alpha * beta) / (alpha + beta));
        } else {
            mu = std::sqrt(Scalar{2.0} / (mu + Scalar{1.0} / mu));
        }

        /*-----------------------------------------------------------------------------
         *  compute relative change of S and Sold
         *-----------------------------------------------------------------------------*/
        Scalar diffSSold, nrmS;
        CHECK_CUSIGNM(cusignm_diffnormFro(n, n, S, Sold, &diffSSold));
        CHECK_CUSIGNM(cusignm_normFro(n, n, S, &nrmS));
        // printf("iter=%d, diffSSold=%e, nrmS=%e, rel. change=%e\n", iter, diffSSold, nrmS, diffSSold / nrmS);

        /*-----------------------------------------------------------------------------
         * stopping criteria
         *-----------------------------------------------------------------------------*/
        // relative change of S and Sold is smaller than tolerance
        if (diffSSold < nrmS * tol) {
            break;
        }

        if (isnan(diffSSold) || isnan(nrmS)) {
            fprintf(stderr, "%s-%s:%d no convergence - NaN detected\n", __func__, __FILE__, __LINE__);
            fflush(stderr);
            ret = -1;
            break;
        }

        // maximum number of iterations reached
        if (iter == maxiter) {
            fprintf(stderr, "%s-%s:%d no convergence - maximum number of iterations reached\n", __func__, __FILE__, __LINE__);
            fflush(stderr);
            ret = -1;
            break;
        }

        /*-----------------------------------------------------------------------------
         * swap S and Sold for the next iteration
         *-----------------------------------------------------------------------------*/
        std::swap(S, Sold);
        iter++;
    }

    /*-----------------------------------------------------------------------------
     * copy S and Sold if necessary
     *-----------------------------------------------------------------------------*/
    if (iter % 2 == 1) {
        CHECK_CUDA(hipMemcpy(Sold, S, sizeof(T) * n * n, hipMemcpyDeviceToDevice));
    }

    /*-----------------------------------------------------------------------------
     * destroy cuBlas and cuSolver handle and params structure
     *-----------------------------------------------------------------------------*/
    CHECK_CUSOLVER(hipsolverDnDestroyParams(params));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    CHECK_CUBLAS(hipblasDestroy(cublasH));

    /*-----------------------------------------------------------------------------
     * return
     *-----------------------------------------------------------------------------*/
    return ret;
}

int cusignm_sNewton(const int n, const float *A, void *d_buffer, void *h_buffer, float *S) {
    return cusignm_Newton(n, A, d_buffer, h_buffer, S);
}

int cusignm_dNewton(const int n, const double *A, void *d_buffer, void *h_buffer, double *S) {
    return cusignm_Newton(n, A, d_buffer, h_buffer, S);
}

int cusignm_cNewton(const int n, const hipComplex *A, void *d_buffer, void *h_buffer, hipComplex *S) {
    return cusignm_Newton(n, A, d_buffer, h_buffer, S);
}

int cusignm_zNewton(const int n, const hipDoubleComplex *A, void *d_buffer, void *h_buffer, hipDoubleComplex *S) {
    return cusignm_Newton(n, A, d_buffer, h_buffer, S);
}
